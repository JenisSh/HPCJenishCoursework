#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__device__ int is_a_match(char *attempt) {
  char password1[] = "AB1111";
  char password2[] = "AB9999";
  char password3[] = "AB6666";
  char password4[] = "AB1966";

  char *j = attempt;
  char *e = attempt;
  char *n = attempt;
  char *i = attempt;
  char *pass1 = password1;
  char *pass2 = password2;
  char *pass3 = password3;
  char *pass4 = password4;

  while(*j == *pass1) {
   if(*j == '\0')
    {
    printf("Found password: %s\n",password1);
      break;
    }

    j++;
    pass1++;
  }
    
  while(*e == *pass2) {
   if(*e == '\0')
    {
    printf("Found password: %s\n",password2);
      break;
}

    e++;
    pass2++;
  }

  while(*n == *pass3) {
   if(*n == '\0')
    {
    printf("Found password: %s\n",password3);
      break;
    }

    n++;
    pass3++;
  }

  while(*i == *pass4) {
   if(*i == '\0')
    {
    printf("Found password: %s\n",password4);
      return 1;
    }

    i++;
    pass4++;
  }
  return 0;

}

__global__ void  kernel() {
char a,b,c,d;
 
  char password[7];
  password[6] = '\0';

int x = blockIdx.x+65;
int y = threadIdx.x+65;
char firstValue = x;
char secondValue = y;
    
password[0] = firstValue;
password[1] = secondValue;
    for(a='0'; a<='9'; a++){
      for(b='0'; b<='9'; b++){
        for(c='0';c<='9';c++){
          for(d='0';d<='9';d++){
            password[2] = a;
            password[3] = b;
            password[4]= c;
            password[5]=d;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



